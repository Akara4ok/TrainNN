#include "hip/hip_runtime.h"
//
// Created by vlad on 4/27/23.
//

#include "Cuda/CudaFunctions.cuh"

namespace GPU {
    __global__ void zeroInit(float* data, int height, int width) {
        const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
        const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
        if ((row < height) && (col < width)) {
            data[row * width + col] = 0;
        }
    }

    __global__ void multiply(float* data, int height, int width, float value) {
        const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
        const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
        if ((row < height) && (col < width)) {
            data[row * width + col] = data[row * width + col] * value;
        }
    }

#ifdef CUDA_STANDARD_SUM

    __global__ void sum(float* result, const float* data, int height, int width, int axis) {
        const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
        const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
        if ((row < height) && (col < width)) {
            switch (axis) {
                case -1:
                    atomicAdd(result, data[row * width + col]);
                    break;
                case 0:
                    atomicAdd(result + row, data[row * width + col]);
                    break;
                case 1:
                    atomicAdd(result + col, data[row * width + col]);
                    break;
                default:
                    break;
            }
        }
    }

#endif
#ifdef CUDA_SHARED_SUM

    __global__ void sum(float* result, const float* data, int height, int width, int axis) {
        __shared__ float dataTile[BLOCK_DIM][BLOCK_DIM];

        const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
        const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

        dataTile[threadIdx.y][threadIdx.x] = 0;
        if ((row < height) && (col < width)) {
            dataTile[threadIdx.y][threadIdx.x] = data[row * width + col];
        }

        __syncthreads();

        if ((row < height) && (col < width)) {
            float subSum = 0;
            if (axis == -1 && threadIdx.x == 0) {
                for (int i = 0; i < blockDim.x; i++) {
                    subSum += dataTile[threadIdx.y][i];
                }
                atomicAdd(result, subSum);
            }
            if (axis == 0 && threadIdx.x == 0) {
                for (int i = 0; i < blockDim.x; i++) {
                    subSum += dataTile[threadIdx.y][i];
                }
                atomicAdd(result + row, subSum);
            }
            if (axis == 1 && threadIdx.y == 0) {
                for (int i = 0; i < blockDim.y; i++) {
                    subSum += dataTile[i][threadIdx.x];
                }
                atomicAdd(result + col, subSum);
            }
        }
    }

#endif

    __global__ void exp(float* result, const float* data, int height, int width) {
        const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
        const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
        if ((row < height) && (col < width)) {
            result[row * width + col] = expf(data[row * width + col]);
        }
    }

    __global__ void log(float* result, const float* data, int height, int width) {
        const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
        const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
        if ((row < height) && (col < width)) {
            result[row * width + col] = logf(data[row * width + col]);
        }
    }

    __global__ void argmax(float* result, const float* data, int height, int width, int axis) {
        const unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;
        if (axis == 0) {
            if (threadId < height) {
                data += threadId * width;
                float maxValue = -1;
                float maxInd = -1;
                for (int i = 0; i < width; i++) {
                    if (data[i] > maxValue) {
                        maxValue = data[i];
                        maxInd = static_cast<float>(i);
                    }
                }
                result[threadId] = maxInd;
            }
        } else if (axis == 1) {
            if (threadId < width) {
                float maxValue = -1;
                float maxInd = -1;
                for (int i = 0; i < height; i++) {
                    if (data[i * width + threadId] > maxValue) {
                        maxValue = data[i * width + threadId];
                        maxInd = static_cast<float>(i);
                    }
                }
                result[threadId] = maxInd;
            }
        }
    }

    __global__ void reciprocal(float* result, const float* data, int height, int width) {
        const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
        const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
        if ((row < height) && (col < width)) {
            result[row * width + col] = 1 / data[row * width + col];
        }
    }

    __global__ void clip(float* result, const float* data, int height, int width,
                         float minBound, float maxBound, float minValueToSet, float maxValueToSet) {
        const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
        const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
        if ((row < height) && (col < width)) {
            result[row * width + col] = data[row * width + col];
            if (data[row * width + col] < minBound) {
                result[row * width + col] = minValueToSet;
            }
            if (data[row * width + col] > maxBound) {
                result[row * width + col] = maxValueToSet;
            }
        }
    }

#ifdef CUDA_STANDARD_TRANSPOSE

    __global__ void transpose(float* result, const float* data, int height, int width) {
        const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
        const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
        if ((row < height) && (col < width)) {
            result[col * height + row] = data[row * width + col];
        }
    }

#endif
#ifdef CUDA_SHARED_TRANSPOSE

    __global__ void transpose(float* result, const float* data, int height, int width) {
        __shared__ float dataTile[BLOCK_DIM][BLOCK_DIM];

        uint col = blockIdx.x * BLOCK_DIM + threadIdx.x;
        uint row = blockIdx.y * BLOCK_DIM + threadIdx.y;

        int totalTiles = BLOCK_DIM / BWL;

        for (int j = 0; j < totalTiles; j++) {
            int currentCol = j * BWL;
            if ((row + currentCol) < height && col < width) {
                dataTile[threadIdx.y + currentCol][threadIdx.x] = data[(row + currentCol) * width + col];
            }
        }

        __syncthreads();

        const uint trow = blockIdx.x * BLOCK_DIM + threadIdx.y;
        const uint tcol = blockIdx.y * BLOCK_DIM + threadIdx.x;

        for (int j = 0; j < totalTiles; j++) {
            int currentCol = j * BWL;
            if (tcol < height && (trow + currentCol) < width) {
                result[(trow + currentCol) * height + tcol] = dataTile[threadIdx.x][threadIdx.y + currentCol];
            }
        }
    }

#endif
#ifdef CUDA_NO_BANK_TRANSPOSE

    __global__ void transpose(float* result, const float* data, int height, int width) {
        __shared__ float dataTile[BLOCK_DIM][BLOCK_DIM + 1];

        uint col = blockIdx.x * BLOCK_DIM + threadIdx.x;
        uint row = blockIdx.y * BLOCK_DIM + threadIdx.y;

        int totalTiles = BLOCK_DIM / BWL;

        for (int j = 0; j < totalTiles; j++) {
            int currentCol = j * BWL;
            if ((row + currentCol) < height && col < width) {
                dataTile[threadIdx.y + currentCol][threadIdx.x] = data[(row + currentCol) * width + col];
            }
        }

        __syncthreads();

        const uint trow = blockIdx.x * BLOCK_DIM + threadIdx.y;
        const uint tcol = blockIdx.y * BLOCK_DIM + threadIdx.x;

        for (int j = 0; j < totalTiles; j++) {
            int currentCol = j * BWL;
            if (tcol < height && (trow + currentCol) < width) {
                result[(trow + currentCol) * height + tcol] = dataTile[threadIdx.x][threadIdx.y + currentCol];
            }
        }
    }

#endif

    __global__ void
    sum(float* result, const float* lhsData, const float* rhsData, int heightLhs, int widthLhs, int heightRhs,
        int widthRhs) {
        const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
        const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
        if ((row < heightLhs) && (col < widthLhs)) {
            const unsigned int rowRhs = heightRhs == heightLhs ? row : 0;
            const unsigned int colRhs = widthRhs == widthLhs ? col : 0;
            result[row * widthLhs + col] = lhsData[row * widthLhs + col] + rhsData[rowRhs * widthRhs + colRhs];
        }
    }

    __global__ void
    subtract(float* result, const float* lhsData, const float* rhsData, int heightLhs, int widthLhs, int heightRhs,
             int widthRhs) {
        const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
        const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
        if ((row < heightLhs) && (col < widthLhs)) {
            const unsigned int rowRhs = heightRhs == heightLhs ? row : 0;
            const unsigned int colRhs = widthRhs == widthLhs ? col : 0;
            result[row * widthLhs + col] = lhsData[row * widthLhs + col] - rhsData[rowRhs * widthRhs + colRhs];
        }
    }

    __global__ void
    elementWiseMultiply(float* result, const float* lhsData, const float* rhsData, int heightLhs, int widthLhs,
                        int heightRhs, int widthRhs) {
        const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
        const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
        if ((row < heightLhs) && (col < widthLhs)) {
            const unsigned int rowRhs = heightRhs == heightLhs ? row : 0;
            const unsigned int colRhs = widthRhs == widthLhs ? col : 0;
            result[row * widthLhs + col] = lhsData[row * widthLhs + col] * rhsData[rowRhs * widthRhs + colRhs];
        }
    }

    __global__ void
    elementWiseDivide(float* result, const float* lhsData, const float* rhsData, int heightLhs, int widthLhs,
                      int heightRhs, int widthRhs) {
        const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
        const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
        if ((row < heightLhs) && (col < widthLhs)) {
            const unsigned int rowRhs = heightRhs == heightLhs ? row : 0;
            const unsigned int colRhs = widthRhs == widthLhs ? col : 0;
            result[row * widthLhs + col] = lhsData[row * widthLhs + col] / rhsData[rowRhs * widthRhs + colRhs];
        }
    }

#ifdef CUDA_STANDARD_MULT

    __global__ void
    multiply(float* result, const float* lhsData, const float* rhsData, int heightLhs, int widthLhs,
             int widthRhs) {
        const uint row = blockIdx.y * blockDim.y + threadIdx.y;
        const uint col = blockIdx.x * blockDim.x + threadIdx.x;
        if ((row < heightLhs) && (col < widthRhs)) {
            float acc = 0.0;
            for (int i = 0; i < widthLhs; i++) {
                acc += lhsData[row * widthLhs + i] * rhsData[i * widthRhs + col];
            }
            result[row * widthRhs + col] = acc;
        }
    }

#endif
#ifdef CUDA_COALESCING_MULT

    __global__ void
    multiply(float* result, const float* lhsData, const float* rhsData, int heightLhs, int widthLhs,
             int widthRhs) {
        const uint row = blockIdx.y * BLOCK_DIM + (threadIdx.x / BLOCK_DIM);
        const uint col = blockIdx.x * BLOCK_DIM + (threadIdx.x % BLOCK_DIM);
        if ((row < heightLhs) && (col < widthRhs)) {
            float acc = 0.0;
            for (int i = 0; i < widthLhs; i++) {
                acc += lhsData[row * widthLhs + i] * rhsData[i * widthRhs + col];
            }
            result[row * widthRhs + col] = acc;
        }
    }

#endif
#ifdef CUDA_SHARED_BLOCK_MULT

    __global__ void
    multiply(float* result, const float* lhsData, const float* rhsData, int heightLhs, int widthLhs,
             int widthRhs) {
        const uint tRow = threadIdx.x / BLOCK_DIM;
        const uint tCol = threadIdx.x % BLOCK_DIM;

        const uint row = blockIdx.y * BLOCK_DIM + tRow;
        const uint col = blockIdx.x * BLOCK_DIM + tCol;

        __shared__ float aTile[BLOCK_DIM][BLOCK_DIM];
        __shared__ float bTile[BLOCK_DIM][BLOCK_DIM];

        float sum = 0;
        const int tiles = (BLOCK_DIM + widthLhs - 1) / BLOCK_DIM;

        uint colLhsOffset = tCol;
        uint rowRhsOffset = tRow;
        for (int tile = 0; tile < tiles; tile++) {
            aTile[tRow][tCol] = 0;
            bTile[tRow][tCol] = 0;
            if (colLhsOffset < widthLhs && row < heightLhs)
                aTile[tRow][tCol] = lhsData[row * widthLhs + colLhsOffset];
            if (rowRhsOffset < widthLhs && col < widthRhs)
                bTile[tRow][tCol] = rhsData[rowRhsOffset * widthRhs + col];

            __syncthreads();

            for (int i = 0; i < BLOCK_DIM; i++) {
                sum += aTile[tRow][i] * bTile[i][tCol];
            }

            __syncthreads();

            colLhsOffset += BLOCK_DIM;
            rowRhsOffset += BLOCK_DIM;
        }
        if ((row < heightLhs) && (col < widthRhs)) {
            result[row * widthRhs + col] = sum;
        }
    }

#endif
#ifdef CUDA_SHARED1D_MULT

    __global__ void
    multiply(float* result, const float* lhsData, const float* rhsData, int heightLhs, int widthLhs,
             int widthRhs) {
        const uint tRow = threadIdx.x / BWR;
        const uint tCol = threadIdx.x % BWR;

        __shared__ float aTile[BHL * BWL];
        __shared__ float bTile[BWL * BWR];

        const uint innerColLhs = threadIdx.x % BWL;
        const uint innerRowLhs = threadIdx.x / BWL;
        const uint innerColRhs = threadIdx.x % BWR;
        const uint innerRowRhs = threadIdx.x / BWR;

        float localRes[THL]{};

        uint rowLhs = blockIdx.y * BHL + innerRowLhs;
        lhsData += rowLhs * widthLhs + innerColLhs;

        uint colRhs = innerColRhs + blockIdx.x * BWR;
        rhsData += innerRowRhs * widthRhs + colRhs;

        uint rowResOffset = tRow * THL + blockIdx.y * BHL;
        uint resCol = tCol + blockIdx.x * BWR;
        result += rowResOffset * widthRhs + resCol;

        uint aTileIdx = innerRowLhs * BWL + innerColLhs;
        uint bTileIdx = innerRowRhs * BWR + innerColRhs;

        for (uint blockId = 0; blockId < widthLhs; blockId += BWL) {
            aTile[aTileIdx] = 0;
            bTile[bTileIdx] = 0;
            if (rowLhs < heightLhs && blockId + innerColLhs < widthLhs) {
                aTile[aTileIdx] = lhsData[0];
            }
            if (blockId + innerRowRhs < widthLhs && colRhs < widthRhs) {
                bTile[bTileIdx] = rhsData[0];
            }

            __syncthreads();

            for (uint subMatrixIndex = 0; subMatrixIndex < BWL; subMatrixIndex++) {
                float temp = bTile[subMatrixIndex * BWR + tCol];
                for (uint resI = 0; resI < THL; resI++) {
                    localRes[resI] += aTile[(tRow * THL + resI) * BWL + subMatrixIndex] * temp;
                }
            }
            __syncthreads();

            lhsData += BWL;
            rhsData += BWL * widthRhs;
        }

        for (uint resI = 0; resI < THL; resI++) {
            if ((rowResOffset + resI) < heightLhs && resCol < widthRhs) {
                result[0] = localRes[resI];
                result += widthRhs;
            }
        }
    }

#endif
#ifdef CUDA_SHARED2D_MULT

    __global__ void
    multiply(float* result, const float* lhsData, const float* rhsData, int heightLhs, int widthLhs,
             int widthRhs) {

        const uint totalResultsBlockTile = BHL * BWR;
        const uint numThreadsBlockTile = totalResultsBlockTile / (THL * TWR);

        const uint tRow = threadIdx.x / (BWR / TWR);
        const uint tCol = threadIdx.x % (BWR / TWR);

        __shared__ float aTile[BHL * BWL];
        __shared__ float bTile[BWL * BWR];

        const uint innerColLhs = threadIdx.x % BWL;
        const uint innerRowLhs = threadIdx.x / BWL;
        const uint innerColRhs = threadIdx.x % BWR;
        const uint innerRowRhs = threadIdx.x / BWR;

        const uint paddingLhs = numThreadsBlockTile / BWL;
        const uint paddingRhs = numThreadsBlockTile / BWR;

        float localRes[THL * TWR]{};

        float regI[THL]{};
        float regJ[TWR]{};

        uint aTileOffset = innerRowLhs * BWL + innerColLhs;
        uint rowLhsOffset = innerRowLhs + blockIdx.y * BHL;
        lhsData += rowLhsOffset * widthLhs + innerColLhs;

        uint bTileOffset = innerRowRhs * BWR + innerColRhs;
        uint colRhsOffset = innerColRhs + blockIdx.x * BWR;
        rhsData += innerRowRhs * widthRhs + colRhsOffset;

        uint rowResOffset = tRow * THL + blockIdx.y * BHL;
        uint rowColOffset = tCol * TWR + blockIdx.x * BWR;
        result += rowResOffset * widthRhs + rowColOffset;

        for (uint blockId = 0; blockId < widthLhs; blockId += BWL) {
            for (uint innerOffset = 0; innerOffset < BHL; innerOffset += paddingLhs) {
                aTile[aTileOffset + innerOffset * BWL] = 0;
                if (rowLhsOffset + innerOffset < heightLhs && blockId + innerColLhs < widthLhs) {
                    aTile[aTileOffset + innerOffset * BWL] = lhsData[innerOffset * widthLhs + blockId];
                }
            }
            for (uint loadOffset = 0; loadOffset < BWL; loadOffset += paddingRhs) {
                bTile[bTileOffset + loadOffset * BWR] = 0;
                if (blockId + innerRowRhs + loadOffset < widthLhs && colRhsOffset < widthRhs) {
                    bTile[bTileOffset + loadOffset * BWR] = rhsData[(blockId + loadOffset) * widthRhs];
                }
            }

            __syncthreads();

            for (uint subMatrixIndex = 0; subMatrixIndex < BWL; ++subMatrixIndex) {
                for (uint i = 0; i < THL; ++i) {
                    regI[i] = aTile[(tRow * THL + i) * BWL + subMatrixIndex];
                }
                for (uint i = 0; i < TWR; ++i) {
                    regJ[i] = bTile[subMatrixIndex * BWR + tCol * TWR + i];
                }
                for (uint resI = 0; resI < THL; ++resI) {
                    for (uint resJ = 0; resJ < TWR; ++resJ) {
                        localRes[resI * TWR + resJ] += regI[resI] * regJ[resJ];
                    }
                }
            }
            __syncthreads();
        }

        for (uint resI = 0; resI < THL; resI++) {
            for (uint resJ = 0; resJ < TWR; resJ++) {
                if (rowResOffset + resI < heightLhs && rowColOffset + resJ < widthRhs) {
                    result[resI * widthRhs + resJ] = localRes[resI * TWR + resJ];
                }
            }
        }
    }

#endif
}