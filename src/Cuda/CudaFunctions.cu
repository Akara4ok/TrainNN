#include "hip/hip_runtime.h"
//
// Created by vlad on 4/27/23.
//

#include "Cuda/CudaFunctions.cuh"
#include "stdio.h"

namespace GPU {
    const int BLOCK_DIM = 32;

    const int BM = 64;
    const int BN = 64;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;

    __global__ void zeroInit(float* data, int height, int width) {
        const unsigned int row = (blockIdx.y * blockDim.y) + threadIdx.y;
        const unsigned int col = (blockIdx.x * blockDim.x) + threadIdx.x;
        if ((row < height) && (col < width)) {
            data[row * width + col] = 0;
        }
    }

    __global__ void multiply(float* data, int height, int width, float value) {
        const unsigned int row = (blockIdx.y * blockDim.y) + threadIdx.y;
        const unsigned int col = (blockIdx.x * blockDim.x) + threadIdx.x;
        if ((row < height) && (col < width)) {
            data[row * width + col] = data[row * width + col] * value;
        }
    }

#ifdef CUDA_STANDARD_SUM
    __global__ void sum(float* result, const float* data, int height, int width, int axis) {
        const unsigned int row = (blockIdx.y * blockDim.y) + threadIdx.y;
        const unsigned int col = (blockIdx.x * blockDim.x) + threadIdx.x;
        if ((row < height) && (col < width)) {
            switch (axis) {
                case -1:
                    atomicAdd(result, data[row * width + col]);
                    break;
                case 0:
                    atomicAdd(result + row, data[row * width + col]);
                    break;
                case 1:
                    atomicAdd(result + col, data[row * width + col]);
                    break;
                default:
                    break;
            }
        }
    }
#endif
#ifdef CUDA_SHARED_SUM
    __global__ void sum(float* result, const float* data, int height, int width, int axis) {
        __shared__ float data_tile[BLOCK_DIM][BLOCK_DIM];

        const unsigned int row = (blockIdx.y * blockDim.y) + threadIdx.y;
        const unsigned int col = (blockIdx.x * blockDim.x) + threadIdx.x;

        data_tile[threadIdx.y][threadIdx.x] = 0;
        if((row < height) && (col < width)){
            data_tile[threadIdx.y][threadIdx.x] = data[row * width + col];
        }

        __syncthreads();

        if((row < height) && (col < width)){
            float subSum = 0;
            if(axis == -1 && threadIdx.x == 0) {
                for (int i = 0; i < blockDim.x; i++){
                    subSum += data_tile[threadIdx.y][i];
                }
                atomicAdd(result, subSum);
            }
            if(axis == 0 && threadIdx.x == 0) {
                for (int i = 0; i < blockDim.x; i++){
                    subSum += data_tile[threadIdx.y][i];
                }
                atomicAdd(result + row, subSum);
            }
            if(axis == 1 && threadIdx.y == 0) {
                for (int i = 0; i < blockDim.y; i++){
                    subSum += data_tile[i][threadIdx.x];
                }
                atomicAdd(result + col, subSum);
            }
        }
    }
#endif

    __global__ void exp(float* result, const float* data, int height, int width) {
        const unsigned int row = (blockIdx.y * blockDim.y) + threadIdx.y;
        const unsigned int col = (blockIdx.x * blockDim.x) + threadIdx.x;
        if ((row < height) && (col < width)) {
            result[row * width + col] = expf(data[row * width + col]);
        }
    }

    __global__ void log(float* result, const float* data, int height, int width) {
        const unsigned int row = (blockIdx.y * blockDim.y) + threadIdx.y;
        const unsigned int col = (blockIdx.x * blockDim.x) + threadIdx.x;
        if ((row < height) && (col < width)) {
            result[row * width + col] = logf(data[row * width + col]);
        }
    }

    __global__ void argmax(float* result, const float* data, int height, int width, int axis) {
        const unsigned int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (axis == 0) {
            if (threadId < height) {
                data += threadId * width;
                float maxValue = -1;
                float maxInd = -1;
                for (int i = 0; i < width; i++) {
                    if (data[i] > maxValue) {
                        maxValue = data[i];
                        maxInd = static_cast<float>(i);
                    }
                }
                result[threadId] = maxInd;
            }
        } else if (axis == 1) {
            if (threadId < width) {
                float maxValue = -1;
                float maxInd = -1;
                for (int i = 0; i < height; i++) {
                    if (data[i * width + threadId] > maxValue) {
                        maxValue = data[i * width + threadId];
                        maxInd = static_cast<float>(i);
                    }
                }
                result[threadId] = maxInd;
            }
        }
    }

    __global__ void reciprocal(float* result, const float* data, int height, int width) {
        const unsigned int row = (blockIdx.y * blockDim.y) + threadIdx.y;
        const unsigned int col = (blockIdx.x * blockDim.x) + threadIdx.x;
        if ((row < height) && (col < width)) {
            result[row * width + col] = 1 / data[row * width + col];
        }
    }

    __global__ void clip(float* result, const float* data, int height, int width,
                         float minBound, float maxBound, float minValueToSet, float maxValueToSet) {
        const unsigned int row = (blockIdx.y * blockDim.y) + threadIdx.y;
        const unsigned int col = (blockIdx.x * blockDim.x) + threadIdx.x;
        if ((row < height) && (col < width)) {
            result[row * width + col] = data[row * width + col];
            if (data[row * width + col] < minBound) {
                result[row * width + col] = minValueToSet;
            }
            if (data[row * width + col] > maxBound) {
                result[row * width + col] = maxValueToSet;
            }
        }
    }

#ifdef CUDA_STANDARD_TRANSPOSE
    __global__ void transpose(float* result, const float* data, int height, int width) {
        const unsigned int row = (blockIdx.y * blockDim.y) + threadIdx.y;
        const unsigned int col = (blockIdx.x * blockDim.x) + threadIdx.x;
        if ((row < height) && (col < width)) {
            result[col * height + row] = data[row * width + col];
        }
    }
#endif

#ifdef CUDA_SHARED_TRANSPOSE
    __global__ void transpose(float *result, const float *data, int height, int width)
    {
        __shared__ float data_tile[BLOCK_DIM][BLOCK_DIM];

        // read the matrix tile into shared memory
            // load one element per thread from device memory (idata) and store it
            // in transposed order in block[][]
        uint col = blockIdx.x * BLOCK_DIM + threadIdx.x;
        uint row = blockIdx.y * BLOCK_DIM + threadIdx.y;
//        if((col < width) && (row < height))
//        {
//            data_tile[threadIdx.y][threadIdx.x] = data[row * width + col];
//        }
        for (int j = 0; j < BLOCK_DIM; j += BK) {
            if ((row + j) < height && col < width)
                data_tile[threadIdx.y + j][threadIdx.x] = data[(row + j) * width + col];
        }


        __syncthreads();

        for (int j = 0; j < BLOCK_DIM; j += BK)
            if((row+j) < height && col < width)
                result[col*height + (row+j)] = data_tile[threadIdx.y+j][threadIdx.x];
//        if((col < width) && (row < height))
//        {
//            result[col * height + row] = data_tile[threadIdx.y][threadIdx.x];
//        }
    }
#endif
#ifdef CUDA_NO_BANK_TRANSPOSE
    __global__ void transpose(float *result, const float *data, int height, int width)
    {
        __shared__ float data_tile[BLOCK_DIM][BLOCK_DIM + 1];

        // read the matrix tile into shared memory
        // load one element per thread from device memory (idata) and store it
        // in transposed order in block[][]
        uint col = blockIdx.x * BLOCK_DIM + threadIdx.x;
        uint row = blockIdx.y * BLOCK_DIM + threadIdx.y;
//        if((col < width) && (row < height))
//        {
//            data_tile[threadIdx.y][threadIdx.x] = data[row * width + col];
//        }
        for (int j = 0; j < BLOCK_DIM; j += BK) {
            if ((row + j) < height && col < width)
                data_tile[threadIdx.y + j][threadIdx.x] = data[(row + j) * width + col];
        }


        __syncthreads();

        for (int j = 0; j < BLOCK_DIM; j += BK)
            if((row+j) < height && col < width)
                result[col*height + (row+j)] = data_tile[threadIdx.y+j][threadIdx.x];
//        if((col < width) && (row < height))
//        {
//            result[col * height + row] = data_tile[threadIdx.y][threadIdx.x];
//        }
    }
#endif

    __global__ void
    sum(float* result, const float* lhsData, const float* rhsData, int heightLhs, int widthLhs, int heightRhs,
        int widthRhs) {
        const unsigned int row = (blockIdx.y * blockDim.y) + threadIdx.y;
        const unsigned int col = (blockIdx.x * blockDim.x) + threadIdx.x;
        if ((row < heightLhs) && (col < widthLhs)) {
            const unsigned int rowRhs = heightRhs == heightLhs ? row : 0;
            const unsigned int colRhs = widthRhs == widthLhs ? col : 0;
            result[row * widthLhs + col] = lhsData[row * widthLhs + col] + rhsData[rowRhs * widthRhs + colRhs];
        }
    }

    __global__ void
    subtract(float* result, const float* lhsData, const float* rhsData, int heightLhs, int widthLhs, int heightRhs,
             int widthRhs) {
        const unsigned int row = (blockIdx.y * blockDim.y) + threadIdx.y;
        const unsigned int col = (blockIdx.x * blockDim.x) + threadIdx.x;
        if ((row < heightLhs) && (col < widthLhs)) {
            const unsigned int rowRhs = heightRhs == heightLhs ? row : 0;
            const unsigned int colRhs = widthRhs == widthLhs ? col : 0;
            result[row * widthLhs + col] = lhsData[row * widthLhs + col] - rhsData[rowRhs * widthRhs + colRhs];
        }
    }

    __global__ void
    elementWiseMultiply(float* result, const float* lhsData, const float* rhsData, int heightLhs, int widthLhs,
                        int heightRhs, int widthRhs) {
        const unsigned int row = (blockIdx.y * blockDim.y) + threadIdx.y;
        const unsigned int col = (blockIdx.x * blockDim.x) + threadIdx.x;
        if ((row < heightLhs) && (col < widthLhs)) {
            const unsigned int rowRhs = heightRhs == heightLhs ? row : 0;
            const unsigned int colRhs = widthRhs == widthLhs ? col : 0;
            result[row * widthLhs + col] = lhsData[row * widthLhs + col] * rhsData[rowRhs * widthRhs + colRhs];
        }
    }

    __global__ void
    elementWiseDivide(float* result, const float* lhsData, const float* rhsData, int heightLhs, int widthLhs,
                      int heightRhs, int widthRhs) {
        const unsigned int row = (blockIdx.y * blockDim.y) + threadIdx.y;
        const unsigned int col = (blockIdx.x * blockDim.x) + threadIdx.x;
        if ((row < heightLhs) && (col < widthLhs)) {
            const unsigned int rowRhs = heightRhs == heightLhs ? row : 0;
            const unsigned int colRhs = widthRhs == widthLhs ? col : 0;
            result[row * widthLhs + col] = lhsData[row * widthLhs + col] / rhsData[rowRhs * widthRhs + colRhs];
        }
    }

#ifdef CUDA_STANDARD_MULT
    __global__ void
    multiply(float* result, const float* lhsData, const float* rhsData, int heightLhs, int widthLhs,
             int widthRhs) {
        const uint row = (blockIdx.y * blockDim.y) + threadIdx.y;
        const uint col = (blockIdx.x * blockDim.x) + threadIdx.x;
        if ((row < heightLhs) && (col < widthRhs)) {
            result[row * widthRhs + col] = 0;
            for (int i = 0; i < widthLhs; i++) {
                result[row * widthRhs + col] += lhsData[row * widthLhs + i] * rhsData[i * widthRhs + col];
            }
        }
    }
#endif
#ifdef CUDA_COALSCING_MULT
    __global__ void
    multiply(float* result, const float* lhsData, const float* rhsData, int heightLhs, int widthLhs,
             int widthRhs) {
        const int BLOCK_DIM = 32;
        const uint row = blockIdx.y * BLOCK_DIM + (threadIdx.x / BLOCK_DIM);
        const uint col = blockIdx.x * BLOCK_DIM + (threadIdx.x % BLOCK_DIM);
        if ((row < heightLhs) && (col < widthRhs)) {
            float acc = 0.0;
            for (int i = 0; i < widthLhs; i++) {
                acc += lhsData[row * widthLhs + i] * rhsData[i * widthRhs + col];
            }
            result[row * widthRhs + col] = acc;
        }
    }
#endif
#ifdef CUDA_SHAREDBLOCK_MULT
    __global__ void
    multiply(float* result, const float* lhsData, const float* rhsData, int heightLhs, int widthLhs,
             int widthRhs) {
        const uint tRow = threadIdx.x / BLOCK_DIM;
        const uint tCol = threadIdx.x % BLOCK_DIM;
        const uint row = blockIdx.y * BLOCK_DIM + tRow;
        const uint col = blockIdx.x * BLOCK_DIM + tCol;
        __shared__ float A_tile[BLOCK_DIM][BLOCK_DIM];
        __shared__ float B_tile[BLOCK_DIM][BLOCK_DIM];
        float acc = 0;
        const int tiles = (BLOCK_DIM + widthLhs - 1) / BLOCK_DIM;
        for (int tile = 0; tile < tiles; tile++){
            A_tile[tRow][tCol] = 0;
            B_tile[tRow][tCol] = 0;
            const uint col_j = (tile * BLOCK_DIM) + tCol;
            const uint row_j = (tile * BLOCK_DIM) + tRow;
            if (col_j < widthLhs && row < heightLhs)
                A_tile[tRow][tCol] = lhsData[row * widthLhs + col_j];
            if(row_j < widthLhs && col < widthRhs)
                B_tile[tRow][tCol] = rhsData[row_j * widthRhs + col];
            __syncthreads();
//            printf("%i\n", threadIdx.x);
            for (int i = 0; i < BLOCK_DIM; i++){
                acc += A_tile[tRow][i] * B_tile[i][tCol];
            }
            __syncthreads();
        }
        if ((row < heightLhs) && (col < widthRhs)) {
            result[row * widthRhs + col] = acc;
        }
    }
#endif
#ifdef CUDA_SHARED1D_MULT
    __global__ void
    multiply(float* result, const float* lhsData, const float* rhsData, int heightLhs, int widthLhs,
             int widthRhs) {
        const uint tRow = threadIdx.x / BN;
        const uint tCol = threadIdx.x % BN;
        __shared__ float A_tile[BM * BK];
        __shared__ float B_tile[BK * BN];

        const uint innerColA = threadIdx.x % BK; // warp-level GMEM coalescing
        const uint innerRowA = threadIdx.x / BK;
        const uint innerColB = threadIdx.x % BN; // warp-level GMEM coalescing
        const uint innerRowB = threadIdx.x / BN;

        float threadResults[TM] = {0.0};

        for (uint bkIdx = 0; bkIdx < widthLhs; bkIdx += BK) {
            // populate the SMEM caches
            A_tile[innerRowA * BK + innerColA] = 0;
            B_tile[innerRowB * BN + innerColB] = 0;
            if((blockIdx.y * BM + innerRowA) < heightLhs && bkIdx + innerColA < widthLhs)
                A_tile[innerRowA * BK + innerColA] = lhsData[(blockIdx.y * BM + innerRowA) * widthLhs + bkIdx + innerColA];
            if(bkIdx + innerRowB < widthLhs && innerColB + blockIdx.x * BN < widthRhs)
                B_tile[innerRowB * BN + innerColB] = rhsData[(bkIdx + innerRowB) * widthRhs + innerColB + blockIdx.x * BN];
            __syncthreads();


            // calculate per-thread results
            for (uint dotIdx = 0; dotIdx < BK; ++dotIdx) {
                // we make the dotproduct loop the outside loop, which facilitates
                // reuse of the Bs entry, which we can cache in a tmp var.
                float tmpB = B_tile[dotIdx * BN + tCol];
                for (uint resIdx = 0; resIdx < TM; ++resIdx) {
                    threadResults[resIdx] +=
                            A_tile[(tRow * TM + resIdx) * BK + dotIdx] * tmpB;
                }
            }
            __syncthreads();
        }

        for (uint resIdx = 0; resIdx < TM; ++resIdx) {
            if ((tRow * TM + resIdx + blockIdx.y * BM) < heightLhs && tCol + blockIdx.x * BN< widthRhs)
                result[(tRow * TM + resIdx + blockIdx.y * BM) * widthRhs + tCol + blockIdx.x * BN] = threadResults[resIdx];
        }
    }
#endif
#ifdef CUDA_SHARED2D_MULT

    __global__ void
    multiply(float* result, const float* lhsData, const float* rhsData, int heightLhs, int widthLhs,
             int widthRhs) {

        const uint totalResultsBlocktile = BM * BN;
        const uint numThreadsBlocktile = totalResultsBlocktile / (TM * TN);

        const uint tRow = threadIdx.x / (BN / TN);
        const uint tCol = threadIdx.x % (BN / TN);
        __shared__ float A_tile[BM * BK];
        __shared__ float B_tile[BK * BN];

        const uint innerColA = threadIdx.x % BK; // warp-level GMEM coalescing
        const uint innerRowA = threadIdx.x / BK;
        const uint innerColB = threadIdx.x % BN; // warp-level GMEM coalescing
        const uint innerRowB = threadIdx.x / BN;

        const uint strideA = numThreadsBlocktile / BK;
        const uint strideB = numThreadsBlocktile / BN;

        float threadResults[TM * TN] = {0.0};

        float regM[TM] = {0.0};
        float regN[TN] = {0.0};

        for (uint bkIdx = 0; bkIdx < widthLhs; bkIdx += BK) {
            // populate the SMEM caches
            for (uint loadOffset = 0; loadOffset < BM; loadOffset += strideA) {
                A_tile[(innerRowA + loadOffset) * BK + innerColA] = 0;
                if (innerRowA + loadOffset + blockIdx.y * BM < heightLhs && bkIdx + innerColA < widthLhs)
                    A_tile[(innerRowA + loadOffset) * BK + innerColA] =
                            lhsData[(innerRowA + loadOffset + blockIdx.y * BM) * widthLhs + bkIdx + innerColA];
            }
            for (uint loadOffset = 0; loadOffset < BK; loadOffset += strideB) {
                B_tile[(innerRowB + loadOffset) * BN + innerColB] = 0;
                if (bkIdx + innerRowB + loadOffset < widthLhs && innerColB + blockIdx.x * BN < widthRhs)
                    B_tile[(innerRowB + loadOffset) * BN + innerColB] =
                            rhsData[(bkIdx + innerRowB + loadOffset) * widthRhs + innerColB + blockIdx.x * BN];
            }

            __syncthreads();

            // calculate per-thread results
            for (uint dotIdx = 0; dotIdx < BK; ++dotIdx) {
                // block into registers
                for (uint i = 0; i < TM; ++i) {
                    regM[i] = A_tile[(tRow * TM + i) * BK + dotIdx];
                }
                for (uint i = 0; i < TN; ++i) {
                    regN[i] = B_tile[dotIdx * BN + tCol * TN + i];
                }
                for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
                    for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
                        threadResults[resIdxM * TN + resIdxN] +=
                                regM[resIdxM] * regN[resIdxN];
                    }
                }
            }
            __syncthreads();
        }

        for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
            for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
                if (tRow * TM + resIdxM + blockIdx.y * BM < heightLhs &&
                    tCol * TN + blockIdx.x * BN + resIdxN < widthRhs)
                    result[(tRow * TM + resIdxM + blockIdx.y * BM) * widthRhs + tCol * TN + blockIdx.x * BN +
                           resIdxN] = threadResults[resIdxM * TN + resIdxN];
            }
        }

    }
#endif
}